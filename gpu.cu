#include "hip/hip_runtime.h"
#include "gpu.h"
#include "common.h"

using namespace std;

// Step 1
// Generate constant variables 
__host__ void variables_initialization(int n){
    //initialize a bunck of stuff here
    GRID_SIZE = size;
    NUM_BINS_PER_DIM = int(ceil(sqrt(n)));
    BIN_SIZE = GRID_SIZE / NUM_BINS_PER_DIM;
    if (BIN_SIZE < cutoff){
        NUM_BINS_PER_DIM = int((GRID_SIZE /  cutoff));
        BIN_SIZE = GRID_SIZE / NUM_BINS_PER_DIM;
    }

    //debug
    if (DEBUG){
        printf("Grid Size is %f \n", GRID_SIZE);
        printf("NUM_BINS_PER_DIM is %d \n", NUM_BINS_PER_DIM);
        printf("BIN_SIZE is %f \n", BIN_SIZE);
        printf("Cutoff %f \n", cutoff);
        printf("dt %f \n", dt);
    }
    
}

// Step 2
// Generate the grid
__host__ Bin* generateGrid(particle_t* particles, int n){
    //initialize the grid with NUM_BINS_PER_DIM^2
    Bin *grid = (Bin *)malloc(NUM_BINS_PER_DIM * NUM_BINS_PER_DIM * sizeof(Bin)); 

    for (int i = 0; i < NUM_BINS_PER_DIM * NUM_BINS_PER_DIM; i++){
        grid[i] = Bin();
    }

    //store the point into the grid
    for (int i = 0; i < n; i++){
        int which_block_x = min((int)(particles[i].x / BIN_SIZE), NUM_BINS_PER_DIM - 1);
        int which_block_y = min((int)(particles[i].y / BIN_SIZE), NUM_BINS_PER_DIM - 1);
        int index = FIND_POS_HOST(which_block_y, which_block_x, NUM_BINS_PER_DIM);
        grid[index].addParticle_host(i);
    }
    return grid;
}

// Step 3
// Push the grid to the gpu
__host__ Bin* push_data_to_device(Bin* grid){
    Bin* bins;
    GPUERRCHK(hipMalloc((void **) &bins, NUM_BINS_PER_DIM * NUM_BINS_PER_DIM * sizeof(Bin)));
    GPUERRCHK(hipMemcpy(bins, grid, NUM_BINS_PER_DIM * NUM_BINS_PER_DIM * sizeof(Bin), hipMemcpyHostToDevice));
    return bins;
}

__host__ particle_t* push_particles_to_device(particle_t* particles, int n){
    particle_t* device_particles;
    GPUERRCHK(hipMalloc((void **) &device_particles, n * sizeof(particle_t)));
    GPUERRCHK(hipMemcpy(device_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice));
    return device_particles;
}

// Step 4
// Generate a dummy buffer for the bins
__host__ Bin* generateRedundantBins(){
    Bin* redundantBins;
    GPUERRCHK(hipMalloc((void **) &redundantBins, NUM_BINS_PER_DIM * NUM_BINS_PER_DIM * sizeof(Bin)));
    return redundantBins;
}

// Step 0
// Clear the redundant bins
__global__ void clear_bins(Bin* redundantBins, int NUM_BINS_PER_DIM){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= NUM_BINS_PER_DIM * NUM_BINS_PER_DIM){ 
        return;
    }
    redundantBins[tid].currentSize = 0;
}


__device__ void compute_force_grid(particle_t* device_particles, Bin* bins, int NUM_BINS_PER_DIM, int tid){

    int i = tid / NUM_BINS_PER_DIM;
    int j = tid % NUM_BINS_PER_DIM;

    int currentIndex = FIND_POS_DEVICE(i, j, NUM_BINS_PER_DIM);
    Bin& currentBin = bins[currentIndex];

    //set acceleration to zero
    for (int k = 0; k < currentBin.currentSize; k++){
        device_particles[currentBin.ids[k]].ax = device_particles[currentBin.ids[k]].ay = 0;
    }
    //check right
    if (j != NUM_BINS_PER_DIM - 1){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i, j+1, NUM_BINS_PER_DIM)]);
    }
    //check diagonal right bot
    if (j != NUM_BINS_PER_DIM - 1 && i != NUM_BINS_PER_DIM - 1){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i+1, j+1, NUM_BINS_PER_DIM)]);
    }
    //check diagonal right top
    if (j != NUM_BINS_PER_DIM - 1 && i != 0){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i-1, j+1, NUM_BINS_PER_DIM)]);
    }
    //check left
    if (j != 0){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i, j-1, NUM_BINS_PER_DIM)]);
    }
    //check diagonal left bot
    if (j != 0 && i != NUM_BINS_PER_DIM - 1){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i+1, j-1, NUM_BINS_PER_DIM)]);
    }
    //check diagonal left top
    if (j != 0 && i != 0){
        compute_force_between_blocks(device_particles,currentBin, bins[FIND_POS_DEVICE(i-1, j-1, NUM_BINS_PER_DIM)]);
    }
    //check top
    if (i != 0){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i-1, j, NUM_BINS_PER_DIM)]);
    }
    //check bot
    if (i != NUM_BINS_PER_DIM - 1){
        compute_force_between_blocks(device_particles, currentBin, bins[FIND_POS_DEVICE(i+1, j, NUM_BINS_PER_DIM)]);
    }
    //compute within itself
    compute_force_between_blocks(device_particles, currentBin, currentBin);
}

__device__ void move_particles(particle_t* device_particles, Bin* bins, Bin* redundantBins, double BIN_SIZE, int NUM_BINS_PER_DIM, double GRID_SIZE, int tid){
    Bin& bin = bins[tid];
    for (int k = 0; k < bin.currentSize; k++){
        move_gpu(&device_particles[bin.ids[k]], GRID_SIZE);
        bin_change(redundantBins, device_particles[bin.ids[k]], bin.ids[k], BIN_SIZE, NUM_BINS_PER_DIM);
    }
}

__global__ void compute_move_particles(particle_t* device_particles, Bin* bins, Bin* redundantBins, double BIN_SIZE, int NUM_BINS_PER_DIM, double GRID_SIZE){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= NUM_BINS_PER_DIM * NUM_BINS_PER_DIM){ 
        return;
    }
    compute_force_grid(device_particles, bins, NUM_BINS_PER_DIM, tid);
    move_particles(device_particles, bins, redundantBins, BIN_SIZE, NUM_BINS_PER_DIM, GRID_SIZE, tid);
}



// Simulation begins
__host__ void simulate_particles(FILE* fsave, particle_t* particles, particle_t* device_particles, Bin* grid, Bin* bins, Bin* redundantBins, int n){

    int num_blocks = (NUM_BINS_PER_DIM * NUM_BINS_PER_DIM + NUM_THREADS - 1) / NUM_THREADS;

    for(int step = 0; step < NSTEPS; step++ ) {

        compute_move_particles <<< num_blocks, NUM_THREADS >>> (device_particles, bins, redundantBins, BIN_SIZE, NUM_BINS_PER_DIM, GRID_SIZE);

        swap(bins, redundantBins);

        clear_bins <<< num_blocks, NUM_THREADS >>> (redundantBins, NUM_BINS_PER_DIM);


        if( fsave && (step%SAVEFREQ) == 0 ) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, device_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
}


int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );


    //generate particles
    set_size( n );
    init_particles( n, particles );
    
    // Step 1 initialize constants and grids
    variables_initialization(n);

    // Step 2 generate grid
    Bin* grid = generateGrid(particles, n);
    
    // Get copy timestamp
    double copy_time = read_timer( );
    
    // Step 3 push the grid to the gpu
    Bin* device_bins = push_data_to_device(grid);

    particle_t* device_particles = push_particles_to_device(particles, n);
    
    // Step 4 generate redundant bins for devie_bins
    // We use it store data and later swap with device bins after each round
    Bin* device_redundant_bins = generateRedundantBins();
    
    // Barrier here
    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    

    //begin simulation
    hipDeviceSynchronize();
    double simulation_time = read_timer( );
    // Simulate Particles
    simulate_particles(fsave, particles, device_particles, grid, device_bins, device_redundant_bins, n);
    simulation_time = read_timer( ) - simulation_time;
    hipDeviceSynchronize();
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    free(grid);
    hipFree(device_bins);
    hipFree(device_redundant_bins);

    if( fsave )
        fclose( fsave );
    
    return 0;
}
